
#include <hip/hip_runtime.h>

// MazeWalker.RayCast
extern "C" __global__  void Ray( float* px, int pxLen0,  float* py, int pyLen0,  double* angles, int anglesLen0,  float* offsets, int offsetsLen0,  double* dists, int distsLen0,  bool* Walls, int WallsLen0,  int* mapHeight, int mapHeightLen0);

// MazeWalker.RayCast
extern "C" __global__  void Ray( float* px, int pxLen0,  float* py, int pyLen0,  double* angles, int anglesLen0,  float* offsets, int offsetsLen0,  double* dists, int distsLen0,  bool* Walls, int WallsLen0,  int* mapHeight, int mapHeightLen0)
{
	int num = blockIdx.x * blockDim.x + threadIdx.x;
	bool flag = num < 1920;
	if (flag)
	{
		int num2 = mapHeight[(0)];
		int num3 = mapHeight[(0)] * 2;
		float num4 = sinf((float)angles[(num)]);
		float num5 = cosf((float)angles[(num)]);
		num4 = ((num4 == 0.0f) ? 0.0001f : num4);
		num5 = ((num5 == 0.0f) ? 0.0001f : num5);
		int num6 = (num5 >= 0.0f) ? 1 : -1;
		int num7 = (num4 >= 0.0f) ? 1 : -1;
		int num8 = (num6 == 1) ? ((int)px[(0)] + 1) : ((int)px[(0)]);
		int num9 = (num7 == 1) ? ((int)py[(0)] + 1) : ((int)py[(0)]);
		float num10 = abs(((float)num8 - px[(0)]) / num5);
		float num11 = abs(1.0f / num5);
		int num12 = (int)(px[(0)] + num10 * num5 + (float)num6 * 0.001f);
		int num13 = (int)(py[(0)] + num10 * num4 + (float)num7 * 0.001f);
		bool flag2 = Walls[(num12 + num13 * mapHeight[(0)])];
		float num14;
		if (flag2)
		{
			num14 = num10;
		}
		else
		{
			bool flag3 = num11 > (float)num3;
			if (flag3)
			{
				num14 = (float)num3;
			}
			else
			{
				num14 = num10;
				int num15 = num12;
				while (num15 > 0 && num15 < num2)
				{
					num14 += num11;
					num12 = (int)(px[(0)] + num14 * num5 + (float)num6 * 0.01f);
					num13 = (int)(py[(0)] + num14 * num4 + (float)num7 * 0.01f);
					bool flag4 = num12 + num13 * mapHeight[(0)] < WallsLen0;
					if (flag4)
					{
						bool flag5 = Walls[(num12 + num13 * mapHeight[(0)])];
						if (flag5)
						{
							break;
						}
					}
					num15 += num6;
				}
			}
		}
		float num16 = abs(((float)num9 - py[(0)]) / num4);
		float num17 = abs(1.0f / num4);
		num12 = (int)(px[(0)] + num16 * num5 + (float)num6 * 0.001f);
		num13 = (int)(py[(0)] + num16 * num4 + (float)num7 * 0.001f);
		bool flag6 = Walls[(num12 + num13 * mapHeight[(0)])];
		float num18;
		if (flag6)
		{
			num18 = num16;
		}
		else
		{
			bool flag7 = num17 > (float)num3;
			if (flag7)
			{
				num18 = (float)num3;
			}
			else
			{
				num18 = num16;
				int num19 = num13;
				while (num19 > 0 && num19 < num2)
				{
					num18 += num17;
					num12 = (int)(px[(0)] + num18 * num5 + (float)num6 * 0.01f);
					num13 = (int)(py[(0)] + num18 * num4 + (float)num7 * 0.01f);
					bool flag8 = num12 + num13 * mapHeight[(0)] < WallsLen0;
					if (flag8)
					{
						bool flag9 = Walls[(num12 + num13 * mapHeight[(0)])];
						if (flag9)
						{
							break;
						}
					}
					num19 += num7;
				}
			}
		}
		bool flag10 = num14 < num18;
		if (flag10)
		{
			offsets[(num)] = abs(py[(0)] + num14 * num4 - (float)((int)(py[(0)] + num14 * num4)));
			dists[(num)] = (double)num14;
		}
		else
		{
			offsets[(num)] = abs(px[(0)] + num18 * num5 - (float)((int)(px[(0)] + num18 * num5)));
			dists[(num)] = (double)num18;
		}
	}
}
