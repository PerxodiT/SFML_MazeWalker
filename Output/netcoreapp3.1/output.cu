#include "hip/hip_runtime.h"

// MazeWalker.RayCast
extern "C" __global__  void Ray( float* px, int pxLen0,  float* py, int pyLen0,  double* angles, int anglesLen0,  float* offsets, int offsetsLen0,  double* dists, int distsLen0,  bool* Walls, int WallsLen0,  int* mapHeight, int mapHeightLen0);

// MazeWalker.RayCast
extern "C" __global__  void Ray( float* px, int pxLen0,  float* py, int pyLen0,  double* angles, int anglesLen0,  float* offsets, int offsetsLen0,  double* dists, int distsLen0,  bool* Walls, int WallsLen0,  int* mapHeight, int mapHeightLen0)
{
	int x = threadIdx.x;
	bool flag = x < 1920;
	if (flag)
	{
		float num = sinf((float)angles[(x)]);
		float num2 = cosf((float)angles[(x)]);
		num = ((num == 0.0f) ? 0.0001f : num);
		num2 = ((num2 == 0.0f) ? 0.0001f : num2);
		float num3 = (float)mapHeight[(0)] * sqrtf(2.0f);
		int num4 = (num2 >= 0.0f) ? 1 : -1;
		int num5 = (num >= 0.0f) ? 1 : -1;
		int num6 = (num4 == 1) ? ((int)px[(0)] + 1) : ((int)px[(0)]);
		int num7 = (num5 == 1) ? ((int)py[(0)] + 1) : ((int)py[(0)]);
		float num8 = ((float)num6 - px[(0)]) / num2;
		num8 = fabsf(num8);
		float num9 = 1.0f / num2;
		num9 = fabsf(num9);
		int num10 = (int)(px[(0)] + num8 * num2 + (float)num4 * 0.001f);
		int num11 = (int)(py[(0)] + num8 * num + (float)num5 * 0.001f);
		bool flag2 = Walls[(num10 + num11 * mapHeight[(0)])];
		float num12;
		if (flag2)
		{
			num12 = num8;
		}
		else
		{
			bool flag3 = num9 > num3;
			if (flag3)
			{
				num12 = num3;
			}
			else
			{
				num12 = num8;
				int num13 = num10;
				while (num13 > 0 && num13 < mapHeight[(0)])
				{
					num12 += num9;
					num10 = (int)(px[(0)] + num12 * num2 + (float)num4 * 0.01f);
					num11 = (int)(py[(0)] + num12 * num + (float)num5 * 0.01f);
					bool flag4 = Walls[(num10 + num11 * mapHeight[(0)])];
					if (flag4)
					{
						break;
					}
					num13 += num4;
				}
			}
		}
		float num14 = ((float)num7 - py[(0)]) / num;
		num14 = fabsf(num14);
		float num15 = 1.0f / num;
		num15 = fabsf(num15);
		num10 = (int)(px[(0)] + num14 * num2 + (float)num4 * 0.001f);
		num11 = (int)(py[(0)] + num14 * num + (float)num5 * 0.001f);
		bool flag5 = Walls[(num10 + num11 * mapHeight[(0)])];
		float num16;
		if (flag5)
		{
			num16 = num14;
		}
		else
		{
			bool flag6 = num15 > num3;
			if (flag6)
			{
				num16 = num3;
			}
			else
			{
				num16 = num14;
				int num17 = num11;
				while (num17 > 0 && num17 < mapHeight[(0)])
				{
					num16 += num15;
					num10 = (int)(px[(0)] + num16 * num2 + (float)num4 * 0.01f);
					num11 = (int)(py[(0)] + num16 * num + (float)num5 * 0.01f);
					bool flag7 = Walls[(num10 + num11 * mapHeight[(0)])];
					if (flag7)
					{
						break;
					}
					num17 += num5;
				}
			}
		}
		bool flag8 = num12 < num16;
		if (flag8)
		{
			offsets[(x)] = (py[(0)] + num12 * num) % 1.0f;
			dists[(x)] = (double)num12;
		}
		else
		{
			offsets[(x)] = (px[(0)] + num16 * num2) % 1.0f;
			dists[(x)] = (double)num16;
		}
	}
}
