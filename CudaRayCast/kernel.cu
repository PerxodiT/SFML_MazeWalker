#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>

using namespace std;
hipError_t RayCastWithCuda(float* px, float* py, double* angles, float* offsets, double* dists, bool* Walls, int* mapHeight);


extern "C" __global__ void Ray(float* px, float* py, double* angles, float* offsets, double* dists, bool* Walls, int* mapHeight)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < 1920)
    {
        int x, y;
        float sin_ang = sin(angles[tid]);
        float cos_ang = cos((float)angles[tid]);
        sin_ang = sin_ang == 0 ? 0.0001F : sin_ang;
        cos_ang = cos_ang == 0 ? 0.0001F : cos_ang;

        float DiagLen = mapHeight[0] * 2;

        int dirX;
        int dirY;

        // Dist fo x's
        // Selecting direction
        dirX = cos_ang >= 0 ? 1 : -1;
        dirY = sin_ang >= 0 ? 1 : -1;
        int mx = dirX == 1 ? (int)px[0] + 1 : (int)px[0];
        int my = dirY == 1 ? (int)py[0] + 1 : (int)py[0];



        float sideDistX = (mx - px[0]) / cos_ang;
        sideDistX = abs(sideDistX);

        float deltaDistX = 1 / cos_ang;
        deltaDistX = abs(deltaDistX);

        float distX;
        x = (int)(px[0] + sideDistX * cos_ang + (dirX * 0.001F));
        y = (int)(py[0] + sideDistX * sin_ang + (dirY * 0.001F));

        if (Walls[x + y * mapHeight[0]])
        {
            distX = sideDistX;
        }
        else if (deltaDistX > DiagLen)
            distX = (float)DiagLen; //When intersects outside the map dist = MaxDist
        else
        {
            distX = sideDistX;
            for (int x2 = x; x2 > 0 && x2 < mapHeight[0]; x2 += dirX)
            {
                distX += deltaDistX;
                x = (int)(px[0] + distX * cos_ang + (dirX * 0.01F));
                y = (int)(py[0] + distX * sin_ang + (dirY * 0.01F));
                if (Walls[x + y * mapHeight[0]])
                {
                    break;
                }

            }
        }


        // Dist fo y's
        float sideDistY = (my - py[0]) / sin_ang;
        sideDistY = abs(sideDistY);

        float deltaDistY = 1 / sin_ang;
        deltaDistY = abs(deltaDistY);

        float distY;
        x = (int)((px[0] + sideDistY * cos_ang) + (dirX * 0.001F));
        y = (int)((py[0] + sideDistY * sin_ang) + (dirY * 0.001F));

        if (Walls[x + y * mapHeight[0]])
        {
            distY = sideDistY;
        }
        else if (deltaDistY > DiagLen)
            distY = (float)DiagLen; //When intersects outside the map dist = MaxDist
        else
        {
            distY = sideDistY;
            for (int y2 = y; y2 > 0 && y2 < mapHeight[0]; y2 += dirY)
            {
                distY += deltaDistY;
                x = (int)((px[0] + distY * cos_ang) + (dirX * 0.01F));
                y = (int)((py[0] + distY * sin_ang) + (dirY * 0.01F));
                if (Walls[x + y * mapHeight[0]])
                {
                    break;
                }
            }
        }
        if (distX < distY)
        {
            offsets[tid] = ((py[0] + distX * sin_ang) - (int)(py[0] + distX * sin_ang));
            dists[tid] = distX;
        }
        else
        {
            offsets[tid] = ((px[0] + distY * cos_ang) - (int)(px[0] + distY * cos_ang));
            dists[tid] = distY;
        }
    }
}

main(void) {

}

hipError_t RayCastWithCuda(float* px, float* py, double* angles, float* offsets, double* dists, bool* Walls, int* mapHeight)
{
    return hipError_t();
}
